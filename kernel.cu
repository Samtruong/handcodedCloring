#include "hip/hip_runtime.h"
#define VERSION 1

/*
Common cpp libraries
*/
#include <stdio.h>
#include <stdlib.h>
#include <set>
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>
#include <cstring>
#include <random>

/*
Vector operations
*/
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

/*
GPU functions
*/


using namespace std;
/*==============================================================================
CSR graph class
==============================================================================*/

// class definition
template <typename ValueT, typename SizeT>
class CSR
{
public:
  CSR(const char *);
  ~CSR() {
    hipFree(csr);
    hipFree(offset);
    hipFree(colors);
    hipFree(rand);
    delete [] adj_matrix;
  };
  SizeT &operator[] (SizeT);
  void print_adj();
  void print_arrays();
  void check_conflict();
  unsigned int nodes;
  unsigned int vertices;
  SizeT* csr;
  SizeT* offset;
  SizeT* colors;
  ValueT* rand;
private:
  SizeT* adj_matrix;
};

// class outline (some functions taken from EEC289Q)

// constructor
template <typename ValueT, typename SizeT>
CSR<ValueT,SizeT>::CSR(const char filename[]) {
  string line;
  ifstream infile(filename);
  if (infile.fail()) {
    cout << "ERROR:  failed to open file" << endl;
    return;
  }

  while (getline(infile, line)) {
    istringstream iss(line);
    if (line.find("%") == string::npos)
      break;
  }

  istringstream iss(line);
  SizeT num_rows, num_cols, num_edges;
  iss >> num_rows >> num_cols >> num_edges;
  this->adj_matrix = new SizeT[num_rows * num_rows];
  memset(this->adj_matrix, 0, num_rows * num_rows * sizeof(bool));
  this->vertices = num_rows;

  while (getline(infile, line)) {
    istringstream iss(line);
    SizeT node1, node2, weight;
    iss >> node1 >> node2 >> weight;

    this->adj_matrix[(node1 - 1) * num_rows + (node2 - 1)] = 1;
    this->adj_matrix[(node2 - 1) * num_rows + (node1 - 1)] = 1;
  }
  infile.close();

  // declare csr and offset
  int csr_length = thrust::reduce(thrust::host,
    this->adj_matrix, this->adj_matrix + this->vertices * this->vertices);

  hipMallocManaged(&(this->csr), csr_length * sizeof(SizeT));
  hipMallocManaged(&(this->offset), this->vertices * sizeof(SizeT));
  // this->csr = new SizeT[csr_length];
  // this->offset = new SizeT[this->vertices];

  // populate csr and offset
  int count = 0;
  for (SizeT v = 0 ; v < this->vertices; v++) {
    this->offset[v] = thrust::reduce(thrust::host,
      this->adj_matrix + (v * this->vertices),
      this->adj_matrix + ((v + 1) * this->vertices) );
    for (SizeT adj = 0; adj < this->vertices; adj++) {
      if (this->adj_matrix[v * this->vertices + adj]) {
        this->csr[count] = adj;
        count++;
      }
    }
  }
  thrust::exclusive_scan(thrust::host, this->offset,
    this->offset + this->vertices, this->offset);

  // create rand array for IS
  hipMallocManaged(&(this->rand), this->vertices * sizeof(ValueT));
  // this->rand = new ValueT[this->vertices];
  random_device rd;
  mt19937 e2(rd());
  e2.seed(1);
  uniform_real_distribution<> dist(0,100);
  for (int v = 0; v < this->vertices; v++) {
    this->rand[v] = dist(e2);
  }

  // allocate memory for colors
  hipMallocManaged(&(this->colors), this->vertices * sizeof(SizeT));
  // this->colors = new SizeT[this->vertices];
  memset(this->colors, -1, this->vertices * sizeof(SizeT));
};


// index overload
template <typename ValueT, typename SizeT>
SizeT & CSR<ValueT,SizeT>::operator[](SizeT idx) {
   return this->adj_matrix[idx];
};

// print first 20 x 20 entries for adj matrix
template <typename ValueT, typename SizeT>
void CSR<ValueT, SizeT>::print_adj() {
  SizeT max_idx = 20;
  if(this->vertices < 20)
    max_idx = this->vertices;
  for (int i = 0; i < max_idx; i++) {
    cout << i << " : [";
    for (int j = 0; j < max_idx; j++) {
      cout << this->adj_matrix[i * this->vertices + j] << ", ";
    }
    cout << "]" << endl;
  }
};

// print first 20 entries for offset and csr
template <typename ValueT, typename SizeT>
void CSR<ValueT, SizeT>::print_arrays() {
  SizeT max_idx = 20;
  if(this->vertices < 20)
    max_idx = this->vertices;
    cout << "CSR: [";
    for (int i = 0; i < max_idx; i++) {
      cout << this->csr[i] << ", ";
    }
    cout << "]" << endl;

    cout << "OFFSET: [";
    for (int i = 0; i < max_idx; i++) {
      cout << this->offset[i] << ", ";
    }
    cout << "]" << endl;

    cout << "COLORS: [";
    for (int i = 0; i < max_idx; i++) {
      cout << this->colors[i] << ", ";
    }
    cout << "]" << endl;

    cout << "RAND: [";
    for (int i = 0; i < max_idx; i++) {
      cout << this->rand[i] << ", ";
    }
    cout << "]" << endl;
};

/*==============================================================================
Check for color conflict
==============================================================================*/
template <typename ValueT, typename SizeT>
void CSR<ValueT, SizeT>::check_conflict() {
  for (SizeT v = 0; v < this->vertices; v++) {
    SizeT start_edge = offset[v];
    SizeT num_neighbors = offset[v + 1] -  offset[v];
    for (SizeT e = start_edge; e < start_edge + num_neighbors; e++) {
      SizeT u = csr[e];
      if ((this->colors[v] == this->colors[u]) && (u != v)) {
        cout << "ERROR: Conflict at node " << v << "and node " << u
        << " at color" << colors[v] << endl;
      }
    }
  }
}

/*==============================================================================
IS color operation - outline taken from Gunrock jpl_color_op
==============================================================================*/
template <typename ValueT, typename SizeT>
#if defined(VERSION) && VERSION == 1
__global__
#else
__device__
#endif
void color_op(SizeT* csr, SizeT* offset, ValueT* rand,
              SizeT* colors, int num_vertices, int iteration) {
  unsigned int v = blockIdx.x * blockDim.x + threadIdx.x;
  if (v < num_vertices) {
    if (colors[v] != -1) return;

    SizeT start_edge = offset[v];
    SizeT num_neighbors = offset[v + 1] -  offset[v];

    bool colormax = true;
    bool colormin = true;
    int color = iteration * 2;

    for (SizeT e = start_edge; e < start_edge + num_neighbors; e++) {
      SizeT u = csr[e];

      if ((colors[u] != -1) && (colors[u] != color + 1) &&
              (colors[u] != color + 2) ||
          (v == u))
        continue;
      if (rand[v] <= rand[u]) colormax = false;
      if (rand[v] >= rand[u]) colormin = false;
    }

    if (colormax) colors[v] = color + 1;
    if (colormin) colors[v] = color + 2;
  }
};

/*==============================================================================
IS color stop condition
==============================================================================*/
template <typename ValueT, typename SizeT>
__host__ __device__
bool stop_condition(SizeT* colors, unsigned int num_vertices) {
#if defined(VERSION) && VERSION == 1
  for (int v = 0; v < num_vertices; v++) {
    if (colors[v] == -1)
      return true;
  }
  return false;
#else
  return false;
#endif
}

/*==============================================================================
IS Kernel function
==============================================================================*/
// template <typename ValueT, typename SizeT>
// __global__
// void ISKernel(SizeT csr, SizeT offset, ValueT rand, SizeT colors, int num_vertices) {
//   int iteration = 0;
//   while (stop_condition(colors)) {
//     color_op(csr, offset, rand, colors, num_vertices, iteration);
//     // TODO: grid wise synchronization
//   }
// };

/*==============================================================================
IS Kernel Driver
==============================================================================*/
// template <typename ValueT, typename SizeT>
// void ISKernelDriver(CSR<ValueT, SizeT> graph) {
//   unsigned int num_threads = 32;
//   unsigned int num_blocks = graph.vertices / num_threads + 1;
//   ISKernel<ValueT, SizeT><<<num_blocks, num_threads>>>
//   (graph.csr,
//    graph.offset,
//    graph.rand,
//    graph.colors,
//    graph.vertices);
// }

/*==============================================================================
Tester - version 1
==============================================================================*/
template <typename ValueT, typename SizeT>
void test_1(bool small) {

  CSR <float, int>  graph = CSR<float, int>("/data-2/topc-datasets/gc-data/offshore/offshore.mtx");
  if (small) {
    CSR <float, int> graph = CSR<float, int>("../gunrock/dataset/small/test_cc.mtx"); }

  int iteration = 0;
  unsigned int num_threads = 32;
  unsigned int num_blocks = graph.vertices / num_threads + 1;

  while (stop_condition<float, int>(graph.colors, graph.vertices)) {
      color_op<float, int><<<num_blocks, num_threads>>>
      (graph.csr,
       graph.offset,
       graph.rand,
       graph.colors,
       graph.vertices,
       iteration);
       hipDeviceSynchronize();
       iteration ++;
  }

  graph.print_adj();
  graph.print_arrays();
  graph.check_conflict();
};

/*==============================================================================
Main function
==============================================================================*/

int main(int argc, char const *argv[]) {
#if defined(VERSION) && VERSION == 1
  cout << "Test small graph" << endl;
  test_1 <float, int> (true);

  cout << "Test large graph" << endl;
  test_1 <float, int> (false);
#endif
  return 0;
}

